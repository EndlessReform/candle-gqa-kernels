
#include <hip/hip_runtime.h>
#include<stdint.h>
#include "hip/hip_fp16.h"
#if __CUDA_ARCH__ >= 800
#include <cuda_bf16.h>
#endif

// Core repeat_kv function template for standard types
template<typename T>
__device__ void repeat_kv(
    const T* key_states, 
    const T* value_states,
    T* repeated_keys,
    T* repeated_values,
    const int n_local_heads,
    const int n_repeats,
    const int seqlen,
    const int head_dim
) {
    int seq_idx = blockIdx.x;
    int head_idx = blockIdx.y;
    int rep_idx = blockIdx.z;
    int dim_idx = threadIdx.x;

    int input_offset = head_idx * seqlen * head_dim + seq_idx * head_dim + dim_idx;
    int expanded_head_idx = head_idx * n_repeats + rep_idx;
    int output_offset = expanded_head_idx * seqlen * head_dim + seq_idx * head_dim + dim_idx;

    repeated_keys[output_offset] = key_states[input_offset];
    repeated_values[output_offset] = value_states[input_offset];
}

// Macro to define repeat_kv kernel for each type
#define REPEAT_KV_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME( \
    const TYPENAME *key_states,      \
    const TYPENAME *value_states,    \
    TYPENAME *repeated_keys,         \
    TYPENAME *repeated_values,       \
    const int n_local_heads,         \
    const int n_repeats,             \
    const int seqlen,                \
    const int head_dim) {            \
    repeat_kv(key_states, value_states, repeated_keys, repeated_values, n_local_heads, n_repeats, seqlen, head_dim); \
}

REPEAT_KV_OP(float, repeat_kv_f32)
REPEAT_KV_OP(double, repeat_kv_f64)
REPEAT_KV_OP(uint8_t, repeat_kv_u8)
REPEAT_KV_OP(uint32_t, repeat_kv_u32)
REPEAT_KV_OP(int64_t, repeat_kv_i64)

#if __CUDA_ARCH__ >= 530
REPEAT_KV_OP(__half, repeat_kv_f16)
#endif

#if __CUDA_ARCH__ >= 800
REPEAT_KV_OP(__nv_bfloat16, repeat_kv_bf16)
#endif